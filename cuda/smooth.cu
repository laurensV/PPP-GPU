#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

float filter[] = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 2.0f, 2.0f, 2.0f, 1.0f, 1.0f, 2.0f, 3.0f, 2.0f, 1.0f, 1.0f, 2.0f, 2.0f, 2.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f};
const unsigned int FILTER_SIZE = 25;

// function to check if there are any cuda errors
void cudaErrorCheck(hipError_t error){
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error: %s\n", hipGetErrorString(error));
		exit(1);
	}
}

// Device code
__global__ void smooth_kernel(const int width, const int height, const int size, const int spectrum, float * filter, unsigned char * inputImage, unsigned char * smoothImage) {
	int gridStride = blockDim.x * gridDim.x;

	for (int pos = blockIdx.x * blockDim.x + threadIdx.x; 
        pos < size; 
        pos += gridStride) {
		darkGrayImage[pos] = ((0.3f * (float)inputImageR[pos]) + (0.59f * (float)inputImageG[pos]) + (0.11f * (float)inputImageB[pos])) * 0.6f + 0.5f;
    }
    /////////////////

    for ( int y = 0; y < height; y++ ) {
		for ( int x = 0; x < width; x++ ) {
			for ( int z = 0; z < spectrum; z++ ) {
				unsigned int filterItem = 0;
				float filterSum = 0.0f;
				float smoothPix = 0.0f;

				for ( int fy = y - 2; fy < y + 3; fy++ ) {
					if ( fy < 0 ) {
						filterItem += 5;
						continue;
					}
					else if ( fy == height ) {
						break;
					}
					
					for ( int fx = x - 2; fx < x + 3; fx++ ) {
						if ( (fx < 0) || (fx >= width) ) {
							filterItem++;
							continue;
						}

						smoothPix += static_cast< float >(inputImage[(z * width * height) + (fy * width) + fx]) * filter[filterItem];
						filterSum += filter[filterItem];
						filterItem++;
					}
				}

				smoothPix /= filterSum;
				smoothImage[(z * width * height) + (y * width) + x] = static_cast< unsigned char >(smoothPix + 0.5f);
			}
		}
	}
}

// Host code
void triangularSmooth(const int width, const int height, const int spectrum, unsigned char * inputImage, unsigned char * smoothImage) {
	// initialize timers
	NSTimer kernelTime = NSTimer("kernelDarker", false, false);	
	NSTimer allocationTime = NSTimer("allocationDarker", false, false);
	NSTimer initTime = NSTimer("initDarker", false, false);
	NSTimer copyDeviceTime = NSTimer("copyDeviceDarker", false, false);
	NSTimer copyHostTime = NSTimer("copyHostDarker", false, false);
	NSTimer freeTime = NSTimer("freeDarker", false, false);
	// init vars
	hipError_t error = hipSuccess;
	unsigned char *inputImageDevice, *smoothImageDevice;
	float * filterDevice;
	int sizeImage = width * height * 3;
	// init call to setup cuda
	initTime.start();
	hipSetDevice(0);
	initTime.stop();

	// allocate images in device memory
	allocationTime.start();
	error = hipMalloc(&inputImageDevice, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&smoothImageDevice, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&filterDevice, FILTER_SIZE * sizeof(float));
	cudaErrorCheck(error);
	allocationTime.stop();

	// Copy image from host to device
	copyDeviceTime.start();
	error = hipMemcpy(inputImageDevice, inputImage, sizeImage, hipMemcpyHostToDevice);
	cudaErrorCheck(error);
	error = hipMemcpy(filterDevice, filter, FILTER_SIZE*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(error);
	copyDeviceTime.stop();

	// number of SM's for GeForce GTX 480
	int numSMs = 32;
	// number of threads per block for GeForce GTX 480
	int threadsPerBlock = 1024;
	// must be a multiple of num SM's for optimal performance
	int numBlocks = 32*numSMs;

	// start the kernel
	kernelTime.start();
	smooth_kernel<<<numBlocks, threadsPerBlock>>>(width, height, sizeImage, spectrum, filterDevice, inputImageDevice, smoothImageDevice);
	cudaErrorCheck(hipGetLastError());
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the result from device to host
	copyHostTime.start();
	error = hipMemcpy(smoothImage, smoothImageDevice, sizeImage, hipMemcpyDeviceToHost);
	cudaErrorCheck(error);
	copyHostTime.stop();
	
	// Free the images in the device memory
	freeTime.start();
	hipFree(inputImageDevice);
	hipFree(filterDevice);
	hipFree(smoothImageDevice);
	freeTime.stop();

	// output times
	cout << fixed << setprecision(6) << "Initalization time: " << initTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Allocation time: " << allocationTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Copy to device time:" << copyDeviceTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Kernel time:" << kernelTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Copy to host time:" << copyHostTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Free time:" << freeTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "GFLOP/s:" << (static_cast< long long unsigned int >(width) * height * 7) / 1000000000.0 / kernelTime.getElapsed() << endl;
	cout << fixed << setprecision(6) << "GB/s:" << (static_cast< long long unsigned int >(width) * height * (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() << endl;
}
