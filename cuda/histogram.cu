#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

const int HISTOGRAM_SIZE = 256;

// function to check if there are any cuda errors
void cudaErrorCheck(hipError_t error){
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error: %s\n", hipGetErrorString(error));
		exit(1);
	}
}

// Device code
__global__ void histogram_kernel(const int width, const int height, const int size, const unsigned char * inputImageR, const unsigned char * inputImageG, const unsigned char * inputImageB, unsigned char * grayImage, unsigned int * histogram) {
	int gridStride = blockDim.x * gridDim.x;

	__shared__ unsigned int sharedHistogram[HISTOGRAM_SIZE];
	if (threadIdx.x < HISTOGRAM_SIZE) {
		sharedHistogram[threadIdx.x] = 0;
	}	
  	__syncthreads();

	for (int pos = blockIdx.x * blockDim.x + threadIdx.x; 
        pos < size; 
        pos += gridStride) {
		grayImage[pos] = ((0.3f * (float)inputImageR[pos]) + (0.59f * (float)inputImageG[pos]) + (0.11f * (float)inputImageB[pos])) + 0.5f;
		atomicAdd(&sharedHistogram[static_cast< unsigned int >(grayImage[pos])],1);
    }
    __syncthreads();
    if (threadIdx.x < HISTOGRAM_SIZE) {
    	atomicAdd(&histogram[threadIdx.x], sharedHistogram[threadIdx.x]);
    }
}


void histogram1D(const int width, const int height, const unsigned char * inputImage, unsigned char * grayImage, unsigned int * histogram, unsigned char * histogramImage) {
	// initialize timers
	NSTimer kernelTime = NSTimer("kernelDarker", false, false);	
	NSTimer allocationTime = NSTimer("allocationDarker", false, false);
	NSTimer initTime = NSTimer("initDarker", false, false);
	NSTimer copyDeviceTime = NSTimer("copyDeviceDarker", false, false);
	NSTimer copyHostTime = NSTimer("copyHostDarker", false, false);
	NSTimer freeTime = NSTimer("freeDarker", false, false);
	// init vars
	hipError_t error = hipSuccess;
	unsigned char *inputImageDeviceR,*inputImageDeviceG, *inputImageDeviceB, *grayImageDevice;
	unsigned int * inputHistogram;
	int sizeImage = width * height;
	// init call to setup cuda
	initTime.start();
	hipSetDevice(0);
	initTime.stop();

	// allocate images in device memory
	allocationTime.start();
	error = hipMalloc(&inputImageDeviceR, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&inputImageDeviceG, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&inputImageDeviceB, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&grayImageDevice, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&inputHistogram, HISTOGRAM_SIZE * sizeof(unsigned int));
	cudaErrorCheck(error);
	allocationTime.stop();

	// Copy image from host to device
	copyDeviceTime.start();
	error = hipMemcpy(inputImageDeviceR, inputImage, sizeImage, hipMemcpyHostToDevice);
	cudaErrorCheck(error);
	error = hipMemcpy(inputImageDeviceG, inputImage+sizeImage, sizeImage, hipMemcpyHostToDevice);
	cudaErrorCheck(error);
	error = hipMemcpy(inputImageDeviceB, inputImage+(sizeImage*2), sizeImage, hipMemcpyHostToDevice);
	cudaErrorCheck(error);
	error = hipMemcpy(inputHistogram, histogram, HISTOGRAM_SIZE * sizeof(unsigned int), hipMemcpyHostToDevice);
	cudaErrorCheck(error);
	copyDeviceTime.stop();

	// number of SM's for GeForce GTX 480
	int numSMs = 32;
	// number of threads per block (minimum is 256)
	int threadsPerBlock = 256;
	// must be a multiple of num SM's for optimal performance
	int numBlocks = 32*numSMs;

	// start the kernel
	kernelTime.start();
	histogram_kernel<<<numBlocks, threadsPerBlock>>>(width, height, sizeImage, inputImageDeviceR, inputImageDeviceG, inputImageDeviceB, grayImageDevice, inputHistogram);
	cudaErrorCheck(hipGetLastError());
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the result from device to host
	copyHostTime.start();
	error = hipMemcpy(grayImage, grayImageDevice, sizeImage, hipMemcpyDeviceToHost);
	cudaErrorCheck(error);	
	error = hipMemcpy(histogram, inputHistogram, HISTOGRAM_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost);
	cudaErrorCheck(error);
	copyHostTime.stop();
	
	// Free the images in the device memory
	freeTime.start();
	hipFree(inputImageDeviceR);
	hipFree(inputImageDeviceG);
	hipFree(inputImageDeviceB);
	hipFree(grayImageDevice);
	hipFree(inputHistogram);
	freeTime.stop();

	// output times
	cout << fixed << setprecision(6) << "Initalization time: " << initTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Allocation time: " << allocationTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Copy to device time:" << copyDeviceTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Kernel time:" << kernelTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Copy to host time:" << copyHostTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "Free time:" << freeTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "GFLOP/s:" << (static_cast< long long unsigned int >(width) * height * 7) / 1000000000.0 / kernelTime.getElapsed() << endl;
	cout << fixed << setprecision(6) << "GB/s:" << (static_cast< long long unsigned int >(width) * height * (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() << endl;
}