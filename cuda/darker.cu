#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

// Device code
__global__ void darkGray_kernel(const int width, const int height, const int size, const unsigned char * inputImage, unsigned char * darkGrayImage) {
	// get position of thread in the 'thread matrix'
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	// Check if we are outside the bounds of the image
	if (x >= width || y >= height) return;

	int pos = (y * width) + x;

	float grayPix = 0.0f;
	float r = static_cast< float >(inputImage[pos]);
	float g = static_cast< float >(inputImage[size + pos]);
	float b = static_cast< float >(inputImage[(2 * size) + pos]);

	grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b));
	grayPix = (grayPix * 0.6f) + 0.5f;

	darkGrayImage[pos] = static_cast< unsigned char >(grayPix);
}

// Host code
void darkGray(const int width, const int height, const unsigned char * inputImage, unsigned char * darkGrayImage) {
	NSTimer kernelTime = NSTimer("kernelDarker", false, false);	
	NSTimer allocationTime = NSTimer("allocationDarker", false, false);
	hipError_t error = hipSuccess;
	int sizeInputImage = width * height * 3;
	int sizedarkGrayImage = width * height;

	// allocate images in device memory
	unsigned char *inputImageDevice, *darkGrayImageDevice;

	allocationTime.start();
	error = hipMalloc(&inputImageDevice, sizeInputImage * sizeof(unsigned char));
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error in hipMalloc input image: %s\n", hipGetErrorString(error));
		exit(1);
	}
	error = hipMalloc(&darkGrayImageDevice, sizedarkGrayImage * sizeof(unsigned char));
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error in hipMalloc gray image: %s\n", hipGetErrorString(error));
		exit(1);
	}	
	allocationTime.stop();
	// Copy image from host to device
	error = hipMemcpy(inputImageDevice, inputImage, sizeInputImage, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error in hipMemcpy image from host to device: %s\n", hipGetErrorString(error));
		exit(1);
	}	
	kernelTime.start();
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(sizeInputImage / threadsPerBlock.x, sizeInputImage / threadsPerBlock.y);
	darkGray_kernel<<<numBlocks, threadsPerBlock>>>(width, height, width*height, inputImageDevice, darkGrayImageDevice);
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error in darkGray_kernel: %s\n", hipGetErrorString(error));
		exit(1);
	}		
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the result from device to host
	error = hipMemcpy(darkGrayImage, darkGrayImageDevice, sizedarkGrayImage, hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error in hipMemcpy result from device to host: %s\n", hipGetErrorString(error));
		exit(1);
	}		
	// Free the images in the device memory
	hipFree(inputImageDevice);
	hipFree(darkGrayImageDevice);
	
	// Time GFLOP/s GB/s
	cout << fixed << setprecision(6) << kernelTime.getElapsed() << setprecision(3) << " " << (static_cast< long long unsigned int >(width) * height * 7) / 1000000000.0 / kernelTime.getElapsed() << " " << (static_cast< long long unsigned int >(width) * height * (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() << endl;
	cout << fixed << setprecision(6) << allocationTime.getElapsed() << setprecision(3) << endl;
}
