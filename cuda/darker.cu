#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

// Device code

void cudaErrorCheck(hipError_t error){
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error: %s\n", hipGetErrorString(error));
		exit(1);
	}
}

__global__ void darkGray_kernel(const int width, const int height, const int size, const unsigned char * inputImageR, const unsigned char * inputImageG, const unsigned char * inputImageB, unsigned char * darkGrayImage) {
	int stride = blockDim.x * gridDim.x;

	for (int pos = blockIdx.x * blockDim.x + threadIdx.x; 
         pos < size; 
         pos += stride) {
		darkGrayImage[pos] = ((0.3f * (float)inputImageR[pos]) + (0.59f * (float)inputImageG[pos]) + (0.11f * (float)inputImageB[pos])) * 0.6f + 0.5f;
    }
}

// Host code
void darkGray(const int width, const int height, const unsigned char * inputImage, unsigned char * darkGrayImage) {
	NSTimer kernelTime = NSTimer("kernelDarker", false, false);	
	NSTimer allocationTime = NSTimer("allocationDarker", false, false);
	NSTimer initTime = NSTimer("initDarker", false, false);
	NSTimer copyDeviceTime = NSTimer("copyDeviceDarker", false, false);
	hipError_t error = hipSuccess;
	unsigned char *inputImageDeviceR,*inputImageDeviceG, *inputImageDeviceB, *darkGrayImageDevice;
	int sizeImage = width * height;

	// init call to setup cuda
	initTime.start();
	hipSetDevice(1);
	initTime.stop();

	// allocate images in device memory
	allocationTime.start();
	error = hipMalloc(&inputImageDeviceR, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&inputImageDeviceG, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&inputImageDeviceB, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&darkGrayImageDevice, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	allocationTime.stop();

	// Copy image from host to device
	copyDeviceTime.start();
	error = hipMemcpy(inputImageDeviceR, inputImage, sizeImage, hipMemcpyHostToDevice);
	error = hipMemcpy(inputImageDeviceG, inputImage+sizeImage, sizeImage, hipMemcpyHostToDevice);
	error = hipMemcpy(inputImageDeviceB, inputImage+(sizeImage*2), sizeImage, hipMemcpyHostToDevice);
	cudaErrorCheck(error);
	copyDeviceTime.stop();

	kernelTime.start();
	//dim3 threadsPerBlock(16, 16);
	//dim3 numBlocks(ceil((float)width / threadsPerBlock.x), ceil((float)height / threadsPerBlock.y));
	int threadsPerBlock = 265;
	//must be a multiple of num SM's for optimal performance
	int numBlocks = 1024;
	darkGray_kernel<<<numBlocks, threadsPerBlock>>>(width, height, sizeImage inputImageDeviceR, inputImageDeviceG, inputImageDeviceB, darkGrayImageDevice);
	cudaErrorCheck(error);
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the result from device to host
	error = hipMemcpy(darkGrayImage, darkGrayImageDevice, sizeImage, hipMemcpyDeviceToHost);
	cudaErrorCheck(error);
	
	// Free the images in the device memory
	hipFree(inputImageDeviceR);
	hipFree(inputImageDeviceG);
	hipFree(inputImageDeviceB);
	hipFree(darkGrayImageDevice);
	
	// Time GFLOP/s GB/s
	cout << fixed << setprecision(6) << "initalization time: " << initTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "allocation time: " << allocationTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "kernel time:" << kernelTime.getElapsed() << setprecision(3) << " " << (static_cast< long long unsigned int >(width) * height * 7) / 1000000000.0 / kernelTime.getElapsed() << " " << (static_cast< long long unsigned int >(width) * height * (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() << endl;
}
