#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

// Device code

void cudaErrorCheck(hipError_t error){
	if (error != hipSuccess) {
		fprintf(stderr, "cuda Error: %s\n", hipGetErrorString(error));
		exit(1);
	}
}

__global__ void darkGray_kernel(const int width, const int height, const unsigned char * inputImageR, const unsigned char * inputImageG, const unsigned char * inputImageB, unsigned char * darkGrayImage) {
	// get position of thread in the 'thread matrix'
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	// Check if we are outside the bounds of the image
	if (x >= width || y >= height) return;

	int pos = (y * width) + x;

	float grayPix = 0.0f;
	float r = static_cast< float >(inputImageR[pos]);
	float g = static_cast< float >(inputImageG[pos]);
	float b = static_cast< float >(inputImageB[pos]);

	grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b));
	grayPix = (grayPix * 0.6f) + 0.5f;

	darkGrayImage[pos] = static_cast< unsigned char >(grayPix);
}

// Host code
void darkGray(const int width, const int height, const unsigned char * inputImage, unsigned char * darkGrayImage) {
	NSTimer kernelTime = NSTimer("kernelDarker", false, false);	
	NSTimer allocationTime = NSTimer("allocationDarker", false, false);
	NSTimer initTime = NSTimer("initDarker", false, false);
	NSTimer copyDeviceTime = NSTimer("copyDeviceDarker", false, false);
	hipError_t error = hipSuccess;
	unsigned char *inputImageDeviceR,*inputImageDeviceG, *inputImageDeviceB, *darkGrayImageDevice;
	int sizeImage = width * height;

	// init call to setup cuda
	initTime.start();
	hipSetDevice(1);
	initTime.stop();

	// allocate images in device memory
	allocationTime.start();
	error = hipMalloc(&inputImageDeviceR, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&inputImageDeviceG, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&inputImageDeviceB, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	error = hipMalloc(&darkGrayImageDevice, sizeImage * sizeof(unsigned char));
	cudaErrorCheck(error);
	allocationTime.stop();

	// Copy image from host to device
	copyDeviceTime.start();
	error = hipMemcpy(inputImageDeviceR, inputImage, sizeImage, hipMemcpyHostToDevice);
	error = hipMemcpy(inputImageDeviceG, inputImage+sizeImage, sizeImage, hipMemcpyHostToDevice);
	error = hipMemcpy(inputImageDeviceB, inputImage+(sizeImage*2), sizeImage, hipMemcpyHostToDevice);
	cudaErrorCheck(error);
	copyDeviceTime.stop();

	kernelTime.start();
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(ceil((float)width / threadsPerBlock.x), ceil((float)height / threadsPerBlock.y));
	darkGray_kernel<<<numBlocks, threadsPerBlock>>>(width, height, inputImageDeviceR, inputImageDeviceG, inputImageDeviceB, darkGrayImageDevice);
	cudaErrorCheck(error);
	
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the result from device to host
	error = hipMemcpy(darkGrayImage, darkGrayImageDevice, sizeImage, hipMemcpyDeviceToHost);
	cudaErrorCheck(error);
	
	// Free the images in the device memory
	hipFree(inputImageDeviceR);
	hipFree(inputImageDeviceG);
	hipFree(inputImageDeviceB);
	hipFree(darkGrayImageDevice);
	
	// Time GFLOP/s GB/s
	cout << fixed << setprecision(6) << "initalization time: " << initTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "allocation time: " << allocationTime.getElapsed() << setprecision(3) << endl;
	cout << fixed << setprecision(6) << "kernel time:" << kernelTime.getElapsed() << setprecision(3) << " " << (static_cast< long long unsigned int >(width) * height * 7) / 1000000000.0 / kernelTime.getElapsed() << " " << (static_cast< long long unsigned int >(width) * height * (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTime.getElapsed() << endl;
}
